#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

__global__ void HL_kernel(const unsigned char* d_data, unsigned int worldWidth, 
                          unsigned int worldHeight, unsigned char* d_resultData){

   size_t index 
   index = threadIdx.x+blockIdx.x*blockDim.x;
   index += blockDim.x*gridDim;
   if (index < worldWidth*worldHeight){
	 size_t x = index % worldWidth;
         size_t y = index / worldHeight;	

 	 size_t x0 = (x + worldWidth -1) % worldWidth;
	 size_t x1 = x;
	 size_t x2 = (x + 1) % worldWidth;

         size_t y0 = ((y + worldHeight -1) % worldHeight) * worldWidth;    
         size_t y1 = y * worldWidth;
         size_t y2 = (( y + 1) % worldHeight) * worldWidth;
         
	 unsigned int aliveCells =  data[x0 + y0] + data[x1 + y0] + data[x2 + y0]
		                     + data[x0 + y1] + data[x2 + y1] + data[x0 + y2] + data[x1 + y2] + data[x2 + y2];
         d_resultData[x1 + y1] = (aliveCells == 3) || (aliveCells == 6 && !d_data[x1+y1])
	 || (aliveCells == 2 && d_data[x1 + y1]) ? 1 : 0;	 
   }                                                    
                          
}                          

// Result from last compute of world.
unsigned char *g_resultData=NULL;

// Current state of world. 
unsigned char *g_data=NULL;

// Current width of world.
size_t g_worldWidth=0;
/// Current height of world.
size_t g_worldHeight=0;

/// Current data length (product of width and height)
size_t g_dataLength=0;  // g_worldWidth * g_worldHeight

static inline void HL_initAllZeros( size_t worldWidth, size_t worldHeight )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;
 
    // calloc init's to all zeros
    hipMallocManaged( &gdata, (g_dataLength, sizeof(unsigned char)));
    hipMemset(gdata, 0, size * sizeof(unsigned char));
    hipMallocManaged(&g_resultData, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(g_resultData, 0, size * sizeof(unsigned char));
}

static inline void HL_initAllOnes( size_t worldWidth, size_t worldHeight )
{
    int i;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged( &gdata, (g_dataLength, sizeof(unsigned char)));
    hipMemset(gdata, 0, size * sizeof(unsigned char));

    // set all rows of world to true
    for( i = 0; i < g_dataLength; i++)
    {
	g_data[i] = 1;
    }
    
    hipMallocManaged(&g_resultData, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(g_resultData, 0, size * sizeof(unsigned char));
     
}

static inline void HL_initOnesInMiddle( size_t worldWidth, size_t worldHeight )
{
    int i;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged( &gdata, (g_dataLength, sizeof(unsigned char)));
    hipMemset(gdata, 0, size * sizeof(unsigned char));

    // set first 1 rows of world to true
    for( i = 10*g_worldWidth; i < 11*g_worldWidth; i++)
    {
	if( (i >= ( 10*g_worldWidth + 10)) && (i < (10*g_worldWidth + 20)))
	{
	    g_data[i] = 1;
	}
    }
    
    hipMallocManaged(&g_resultData, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(g_resultData, 0, size * sizeof(unsigned char)); 
}

static inline void HL_initOnesAtCorners( size_t worldWidth, size_t worldHeight )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged( &gdata, (g_dataLength, sizeof(unsigned char)));
    hipMemset(gdata, 0, size * sizeof(unsigned char));

    g_data[0] = 1; // upper left
    g_data[worldWidth-1]=1; // upper right
    g_data[(worldHeight * (worldWidth-1))]=1; // lower left
    g_data[(worldHeight * (worldWidth-1)) + worldWidth-1]=1; // lower right
    
    hipMallocManaged(&g_resultData, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(g_resultData, 0, size * sizeof(unsigned char)); 
}

static inline void HL_initSpinnerAtCorner( size_t worldWidth, size_t worldHeight )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged(&gdata, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(gdata, 0, size * sizeof(unsigned char)); 

    g_data[0] = 1; // upper left
    g_data[1] = 1; // upper left +1
    g_data[worldWidth-1]=1; // upper right
    
    hipMallocManaged(&g_resultData, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(g_resultData, 0, size * sizeof(unsigned char));  
}

static inline void HL_initReplicator( size_t worldWidth, size_t worldHeight )
{
    size_t x, y;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged(&gdata, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(gdata, 0, size * sizeof(unsigned char)); 

    x = worldWidth/2;
    y = worldHeight/2;
    
    g_data[x + y*worldWidth + 1] = 1; 
    g_data[x + y*worldWidth + 2] = 1;
    g_data[x + y*worldWidth + 3] = 1;
    g_data[x + (y+1)*worldWidth] = 1;
    g_data[x + (y+2)*worldWidth] = 1;
    g_data[x + (y+3)*worldWidth] = 1; 
    
    hipMallocManaged(&g_resultData, ( g_dataLength, sizeof(unsigned char))); 
    hipMemset(g_resultData, 0, size * sizeof(unsigned char)); 
}

static inline void HL_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight )
{
    switch(pattern)
    {
    case 0:
	HL_initAllZeros( worldWidth, worldHeight );
	break;
	
    case 1:
	HL_initAllOnes( worldWidth, worldHeight );
	break;
	
    case 2:
	HL_initOnesInMiddle( worldWidth, worldHeight );
	break;
	
    case 3:
	HL_initOnesAtCorners( worldWidth, worldHeight );
	break;

    case 4:
	HL_initSpinnerAtCorner( worldWidth, worldHeight );
	break;

    case 5:
	HL_initReplicator( worldWidth, worldHeight );
	break;
	
    default:
	printf("Pattern %u has not been implemented \n", pattern);
	exit(-1);
    }
}

static inline void HL_swap( unsigned char **pA, unsigned char **pB)
{
  unsigned char *temp = *pA;
  *pA = *pB;
  *pB = temp;
}
 

// Don't Modify this function or your submitty autograding will not work
static inline void HL_printWorld(size_t iteration)
{
    int i, j;

    printf("Print World - Iteration %lu \n", iteration);
    
    for( i = 0; i < g_worldHeight; i++)
    {
	printf("Row %2d: ", i);
	for( j = 0; j < g_worldWidth; j++)
	{
	    printf("%u ", (unsigned int)g_data[(i*g_worldWidth) + j]);
	}
	printf("\n");
    }

    printf("\n\n");
}

bool HL_kernelLaunch(unsigned char** d_data, unsigned char** d_resultData, size_t worldWidth, size_t worldHeight, size_t iterationsCount, ushort threadsCount)
{
    int blockSize, gridSize;
    unsigned char *d_data, *d_resultData;
    size_t size = worldWidth * worldHeight * sizeof(unsigned char);
    
    hipMallocManaged(&d_data, size);
    hipMallocManaged(&d_resultData, size);
    
   
    size_t blockGridSize = (worldWidth * worldHeight + threadsCount - 1) / threadsCount;
    
    HL_kernel<<<blockGridSize, threadsCount>>>(d_data, worldWidth, worldHeight, d_resultData);
    
    hipDeviceSynchronize();
    HL_swap(&d_data, &d_resultData);
 
    hipFree(d_data);
    hipFree(d_resultData);
    
    return true;
}

int main(int argc, char *argv[])
{
    unsigned int pattern = 0;
    unsigned int worldSize = 0;
    unsigned int iterations = 0;
    unsigned int threadBlockSize = 0;
    
    unsigned char *d_data, *d_resultData;
    

    printf("This is the HighLife running in parallel on a GPU.\n");

    if( argc != 5)
    {
	printf("HighLife requires 4 arguments, 1st is pattern number, 2nd the sq size of the world, 3rd is the number of iterations, and 4th is thread blocksize,  e.g. ./highlife 4 64 2 32 \n");
	exit(-1);
    }
    
    pattern = atoi(argv[1]);
    worldSize = atoi(argv[2]);
    iterations = atoi(argv[3]);
    threadBlockSize = atoi(argv[4]);  
    
    size_t size = worldSize * worldSize * sizeof(unsigned char)
    hipMallocManaged(&d_data, size);
    hipMallocManaged(&d_resultData, size); 
    HL_initMaster(pattern, worldSize, worldSize);

    size_t  blockGridSize = (worldSize*worldSize + threadBlockSize - 1) / threadBlockSize;

    printf("AFTER INIT IS............\n");
    HL_printWorld(0);
   for (size_t i; i < iterations; i++){  
  	 HL_KernelLaunch(d_data,d_resultData,worldSize,worldSize,i,threadBlockSize);
   }
  
    printf("######################### FINAL WORLD IS ###############################\n");
    HL_printWorld(iterations);

    hipFree(d_data);
    hipFree(d_resultData);
    
    return true;
}
